#include "hip/hip_runtime.h"
#include "filter.cuh"

#include <math.h>
#include "third_party/hip/hip_vector_types.h"


int idxToKernel(int d, int kernelSize){
    return d - kernelSize/2;
}

int2 idxToKernel(int2 d, int kernelSize){
    return {idxToKernel(d.x, kernelSize), idxToKernel(d.y, kernelSize)};
}

float gaussian(float2 p, float sigma){
    return exp(-(p.x*p.x+p.y*p.y)/(2*sigma*sigma));
}

float gaussian(float3 p, float sigma){
    return exp(-(p.x*p.x+p.y*p.y+p.z*p.z)/(2*sigma*sigma));
}

float3 log10(float3 f){
    return {log10(f.x), log10(f.y), log10(f.z)};
}

float3 snrCPU(float3* original, float3* noisy, int2 size){
    float3 oriSum = {0,0,0};
    float3 distSum = {0,0,0};
    int2 pos = {0,0};
    do {
        oriSum += original[index(pos, size)]*original[index(pos, size)];
        distSum += (original[index(pos, size)] - noisy[index(pos, size)])*(original[index(pos, size)] - noisy[index(pos, size)]);

    } while(advanceIterator(pos, size));

    return 10.0 * log10(oriSum/distSum);
}


inline bool advanceIterator(int2& pos, int2 size) {
    pos.y++;
    if (pos.y < size.y)
        return true;

    pos.y = 0;
    pos.x++;
    if (pos.x < size.x)
        return true;

    return false;
}

void waveletfilterCPU(float3* in, float3* out, float3* albedo, float3* normal, int2 shape,
    int kerSize, int depth, float sigmaSpace, float sigmaColor, float sigmaAlbedo, float sigmaNormal)
{    
    float3* buffer[2] = {in, out};    
    for(int i = 0; i < depth; i++){
        int2 pos = {0,0};
        do {
            out[index(pos, shape)] =
                waveletfilterPixel(pos, buffer[i%2], buffer[(i+1)%2], albedo, normal, shape, kerSize, 1<<i, sigmaSpace, sigmaColor, sigmaAlbedo, sigmaNormal);
        } while(advanceIterator(pos, shape));
    }

    in = buffer[depth%2];
    out = buffer[(depth+1)%2];
}

void waveletfilterGPU(float3* in, float3* out, float3* albedo, float3* normal, int2 size,
    int kerSize, float sigmaSpace, float sigmaColor, float sigmaAlbedo, float sigmaNormal)
{
    float* buffer[2];
    
    //hipMalloc(&buffer[0], );

}

__global__ void waveletKernel(float3* in, float3* out, float3* albedo, float3* normal, int2 size,
    int kerSize, int offset, float sigmaSpace, float sigmaColor, float sigmaAlbedo, float sigmaNormal){

}

float3 waveletfilterPixel(int2 pos, float3* in, float3* out, float3* albedo, float3* normal, int2 size,
    int kerSize, int offset, float sigmaSpace, float sigmaColor, float sigmaAlbedo, float sigmaNormal)
{    
    float3 acum = {0, 0, 0};
    float normFactor = 0;
    int halfSize = kerSize/2;
    float h[3] = {3.0/8.0, 1.0/4.0, 1.0/16.0}; // Coefs of Pascal TriaNGLE
    int2 d, n;

    //float lp = 0.2126 * in[pos].x + 0.7152 * in[p].y + 0.0722 * in[p].z;

    for(d.x = -halfSize; d.x <= halfSize; d.x++){
        for(d.y = -halfSize; d.y <= halfSize; d.y++){
            n.x = pos.x + d.x * offset;
            n.y = pos.y + d.y * offset;

            if( n.x >= 0 && n.x < size.x &&
                n.y >= 0 && n.y < size.y ){
                float3 dcol    = in[index(n, size)] - in[index(pos, size)];
                float3 dAlbedo = albedo[index(n, size)] - albedo[index(pos, size)];
                //float3 dNormal = 1.0 - normal[n]*normal[pos];
                float wWavelet = h[abs(d.x)] * h[abs(d.y)];
                

                float w =
                    wWavelet 
                    //* gaussian(make_float2(d*offset), sigmaSpace) * // Simplify using exp(a) * exp(b) = exp(a + b)
                    * gaussian(dcol, sigmaColor);
                    //* gaussian(dAlbedo, sigmaAlbedo);

                acum += in[index(n, size)] * w;
                normFactor += w;
            }
        }
    }
    acum /= normFactor;
    return acum;

}