#include "image.cuh"

#include <string>
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "third_party/stb_image.h"
#include "third_party/stb_image_write.h"

#include "third_party/hip/hip_vector_types.h"

#include <stdexcept>


Image::Image(CPUMat3D<float> fmat){
    mat = CPUMat3D<uchar>(fmat.size);
    for(int i = 0; i < fmat.totalSize(); i++)
        mat.data[i] = static_cast<uchar>(fmat.data[i]*255);
    stbi_allocated = false;
}


Image::Image(std::string filename){
    mat.data = (uchar*) stbi_load(filename.c_str(), &(mat.size.x), &(mat.size.y), &(mat.size.z), 0);
    stbi_allocated = true;
}

Image::~Image(){
    if(stbi_allocated)
        close();
}

bool Image::close(){
    stbi_image_free(mat.data);
    return true;
}

bool Image::save(std::string filename){
    return stbi_write_png(filename.c_str(), mat.size.x, mat.size.y, mat.size.z, mat.data, mat.size.x * mat.size.z);
}

CPUMat3D<float> fmatFromImage(const Image& img){
    CPUMat3D<float> out(img.mat.size);
    for(int i = 0; i < img.mat.totalSize(); i++)
        out.data[i] = static_cast<float>(img.mat.data[i])/255;

    return out;
}