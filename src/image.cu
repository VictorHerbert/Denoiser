#include "image.cuh"

#include <string>
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "third_party/stb_image.h"
#include "third_party/stb_image_write.h"

#include "third_party/hip/hip_vector_types.h"

#include <stdexcept>


bool Image::read(std::string filename){
    mat.data = (uchar*) stbi_load(filename.c_str(), &(mat.size.x), &(mat.size.y), &(mat.size.z), 0);
    return true;
}

bool Image::close(){
    stbi_image_free(mat.data);
    return true;
}

bool Image::save(std::string filename){
    return stbi_write_png(filename.c_str(), mat.size.x, mat.size.y, mat.size.z, mat.data, mat.size.x * mat.size.z);
}
