
#include <hip/hip_runtime.h>
int index(int x, int y, int2 shape){
    return x * shape.y + y;
}

int index(int2 p, int2 shape){
    return p.x * shape.y + p.y;
}

bool advanceIterator(int2& pos, int2 shape){
    pos.y++;
    if (pos.y < shape.y)
        return true;

    pos.y = 0;
    pos.x++;
    if (pos.x < shape.x)
        return true;

    return false;
}

int totalSize(int2 shape){
    return shape.x * shape.y;
}

int totalSize(int3 shape){
    return shape.x * shape.y * shape.z;
}